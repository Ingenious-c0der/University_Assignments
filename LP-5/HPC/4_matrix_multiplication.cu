// assignment for matrix multiplication, parallel and sequential comparison

#include <hip/hip_runtime.h>
#include <iostream>

 __global__ void multiply(int *A, int *B, int *C, int size)
{

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < size && col < size)
    {
        int sum = 0;
        for (int i = 0; i < size; i++)
        {
            sum += A[row * size + i] * B[i * size + col];
        }
        C[row * size + col] = sum;
    }
}

void print(int *mat, int size)
{
        for (int row = 0; row < size; row++) {
        for (int col = 0; col < size; col++) {
            std::cout << mat[row * size + col] << " ";
        }
        std::cout << '\n';
    }
    std::cout << '\n';

}

void initialize(int *mat, int size)
{
    for (int i = 0; i < size; i++)
    {
        mat[i] = rand() % (21) - 10;
    }
}

int main()
{
    int N = 2;
    int mat_size = N * N;
    size_t mat_bytes = mat_size * sizeof(int);
    int num_THREADS = 2;
    int threadsPerBlock = N / num_THREADS;
    dim3 threads(num_THREADS, num_THREADS);
    dim3 blocks(2,2);
    int *A, *B, *C;
    A = new int[mat_size];
    B = new int[mat_size];
    C = new int[mat_size];

    initialize(A, mat_size);
    initialize(B, mat_size);
    std::cout << "Matrix A : " <<std::endl;
    print(A, N);
    std::cout << std::endl;

    std::cout << "Matrix B : " << std::endl;
    print(B, N);
    std ::cout << std ::endl;

    int *X, *Y, *Z;
    hipMalloc(&X, mat_bytes);
    hipMalloc(&Y, mat_bytes);
    hipMalloc(&Z, mat_bytes);

    hipMemcpy(X, A, mat_bytes, hipMemcpyHostToDevice);
    hipMemcpy(Y, B, mat_bytes, hipMemcpyHostToDevice);

    multiply<<< blocks, threads>>>(X, Y, Z, mat_size);

    hipMemcpy(C, Z, mat_bytes, hipMemcpyDeviceToHost);
    std::cout << "matrix C : " <<std::endl;
    print(C, N);



    delete[] A ;
    delete[] B;
    delete[] C;

    hipFree(X); 
    hipFree(Y); 
    hipFree(Z); 

    return 0; 
}